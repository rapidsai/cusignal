
#include <hip/hip_runtime.h>
// Copyright (c) 2019-2020, NVIDIA CORPORATION.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

///////////////////////////////////////////////////////////////////////////////
//                                SOSFILT                                    //
///////////////////////////////////////////////////////////////////////////////

constexpr int sos_width = 6;

template<typename T>
__device__ void _cupy_sosfilt( const int n_signals,
                               const int n_samples,
                               const int n_sections,
                               const int zi_width,
                               const T *__restrict__ sos,
                               const T *__restrict__ zi,
                               T *__restrict__ x_in,
                               T *s_buffer ) {

    T *s_out { s_buffer };
    T *s_zi { reinterpret_cast<T *>( &s_out[n_sections] ) };
    T *s_sos { reinterpret_cast<T *>( &s_zi[n_sections * zi_width] ) };

    const int tx { static_cast<int>( threadIdx.x ) };
    const int ty { static_cast<int>( blockIdx.y * blockDim.y + threadIdx.y ) };

    // Reset shared memory
    s_out[tx] = 0;

    // Load zi
    for ( int i = 0; i < zi_width; i++ ) {
        s_zi[tx * zi_width + i] = zi[ty * n_sections * zi_width + tx * zi_width + i];
    }

    // Load SOS
    // b is in s_sos[tx * sos_width + [0-2]]
    // a is in s_sos[tx * sos_width + [3-5]]
#pragma unroll sos_width
    for ( int i = 0; i < sos_width; i++ ) {
        s_sos[tx * sos_width + i] = sos[tx * sos_width + i];
    }

    __syncthreads( );

    const int load_size { n_sections - 1 };
    const int unload_size { n_samples - load_size };

    T temp {};
    T x_n {};

    if ( ty < n_signals ) {
        // Loading phase
        for ( int n = 0; n < load_size; n++ ) {
            if ( tx == 0 ) {
                x_n = x_in[ty * n_samples + n];
            } else {
                x_n = s_out[tx - 1];
            }

            // Use direct II transposed structure
            temp = s_sos[tx * sos_width + 0] * x_n + s_zi[tx * zi_width + 0];

            s_zi[tx * zi_width + 0] =
                s_sos[tx * sos_width + 1] * x_n - s_sos[tx * sos_width + 4] * temp + s_zi[tx * zi_width + 1];

            s_zi[tx * zi_width + 1] = s_sos[tx * sos_width + 2] * x_n - s_sos[tx * sos_width + 5] * temp;

            s_out[tx] = temp;

            __syncthreads( );
        }

        // Processing phase
        for ( int n = load_size; n < n_samples; n++ ) {
            if ( tx == 0 ) {
                x_n = x_in[ty * n_samples + n];
            } else {
                x_n = s_out[tx - 1];
            }

            // Use direct II transposed structure
            temp = s_sos[tx * sos_width + 0] * x_n + s_zi[tx * zi_width + 0];

            s_zi[tx * zi_width + 0] =
                s_sos[tx * sos_width + 1] * x_n - s_sos[tx * sos_width + 4] * temp + s_zi[tx * zi_width + 1];

            s_zi[tx * zi_width + 1] = s_sos[tx * sos_width + 2] * x_n - s_sos[tx * sos_width + 5] * temp;

            if ( tx < load_size ) {
                s_out[tx] = temp;
            } else {
                x_in[ty * n_samples + ( n - load_size )] = temp;
            }

            __syncthreads( );
        }

        // Unloading phase
        for ( int n = 0; n < n_sections; n++ ) {
            // retire threads that are less than n
            if ( tx > n ) {
                x_n = s_out[tx - 1];

                // Use direct II transposed structure
                temp = s_sos[tx * sos_width + 0] * x_n + s_zi[tx * zi_width + 0];

                s_zi[tx * zi_width + 0] =
                    s_sos[tx * sos_width + 1] * x_n - s_sos[tx * sos_width + 4] * temp + s_zi[tx * zi_width + 1];

                s_zi[tx * zi_width + 1] = s_sos[tx * sos_width + 2] * x_n - s_sos[tx * sos_width + 5] * temp;

                if ( tx < load_size ) {
                    s_out[tx] = temp;
                } else {
                    x_in[ty * n_samples + ( n + unload_size )] = temp;
                }
                __syncthreads( );
            }
        }
    }
}

extern "C" __global__ void __launch_bounds__( 1024 ) _cupy_sosfilt_float32( const int n_signals,
                                                                            const int n_samples,
                                                                            const int n_sections,
                                                                            const int zi_width,
                                                                            const float *__restrict__ sos,
                                                                            const float *__restrict__ zi,
                                                                            float *__restrict__ x_in ) {

    extern __shared__ float s_buffer_f[];

    _cupy_sosfilt<float>( n_signals, n_samples, n_sections, zi_width, sos, zi, x_in, s_buffer_f );
}

extern "C" __global__ void __launch_bounds__( 1024 ) _cupy_sosfilt_float64( const int n_signals,
                                                                            const int n_samples,
                                                                            const int n_sections,
                                                                            const int zi_width,
                                                                            const double *__restrict__ sos,
                                                                            const double *__restrict__ zi,
                                                                            double *__restrict__ x_in ) {

    extern __shared__ double s_buffer_d[];

    _cupy_sosfilt<double>( n_signals, n_samples, n_sections, zi_width, sos, zi, x_in, s_buffer_d );
}
