
#include <hip/hip_runtime.h>
// Copyright (c) 2019-2020, NVIDIA CORPORATION.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

///////////////////////////////////////////////////////////////////////////////
//                                SOSFILT                                    //
///////////////////////////////////////////////////////////////////////////////

constexpr int sos_width = 6;

template<typename T>
__device__ void _cupy_sosfilt( const int n_signals,
                               const int n_samples,
                               const int n_sections,
                               const int zi_width,
                               const T *__restrict__ sos,
                               const T *__restrict__ zi,
                               T *__restrict__ x_in,
                               T *s_buffer ) {

    T *s_out { s_buffer };
    T *s_sos { reinterpret_cast<T *>( &s_out[n_sections] ) };

    const int tx { static_cast<int>( threadIdx.x ) };
    const int bx { static_cast<int>( blockIdx.x ) };

    // Reset shared memory
    s_out[tx] = 0;

    // Load SOS
    // b is in s_sos[tx * sos_width + [0-2]]
    // a is in s_sos[tx * sos_width + [3-5]]
#pragma unroll sos_width
    for ( int i = 0; i < sos_width; i++ ) {
        s_sos[tx * sos_width + i] = sos[tx * sos_width + i];
    }

    // __syncthreads( );

    T zi0 = zi[bx * n_sections * zi_width + tx * zi_width + 0];
    T zi1 = zi[bx * n_sections * zi_width + tx * zi_width + 1];

    const int load_size { n_sections - 1 };
    const int unload_size { n_samples - load_size };

    T temp {};
    T x_n {};

    if ( bx < n_signals ) {
        // Loading phase
        for ( int n = 0; n < load_size; n++ ) {
            __syncthreads( );
            if ( tx == 0 ) {
                x_n = x_in[bx * n_samples + n];
            } else {
                x_n = s_out[tx - 1];
            }

            // Use direct II transposed structure
            temp = s_sos[tx * sos_width + 0] * x_n + zi0;
            zi0  = s_sos[tx * sos_width + 1] * x_n - s_sos[tx * sos_width + 4] * temp + zi1;
            zi1  = s_sos[tx * sos_width + 2] * x_n - s_sos[tx * sos_width + 5] * temp;

            s_out[tx] = temp;
        }

        // Processing phase
        for ( int n = load_size; n < n_samples; n++ ) {
            __syncthreads( );
            if ( tx == 0 ) {
                x_n = x_in[bx * n_samples + n];
            } else {
                x_n = s_out[tx - 1];
            }

            // Use direct II transposed structure
            temp = s_sos[tx * sos_width + 0] * x_n + zi0;
            zi0  = s_sos[tx * sos_width + 1] * x_n - s_sos[tx * sos_width + 4] * temp + zi1;
            zi1  = s_sos[tx * sos_width + 2] * x_n - s_sos[tx * sos_width + 5] * temp;

            if ( tx < load_size ) {
                s_out[tx] = temp;
            } else {
                x_in[bx * n_samples + ( n - load_size )] = temp;
            }
        }

        // Unloading phase
        for ( int n = 0; n < n_sections; n++ ) {
            __syncthreads( );
            // retire threads that are less than n
            if ( tx > n ) {
                x_n = s_out[tx - 1];

                // Use direct II transposed structure
                temp = s_sos[tx * sos_width + 0] * x_n + zi0;
                zi0  = s_sos[tx * sos_width + 1] * x_n - s_sos[tx * sos_width + 4] * temp + zi1;
                zi1  = s_sos[tx * sos_width + 2] * x_n - s_sos[tx * sos_width + 5] * temp;

                if ( tx < load_size ) {
                    s_out[tx] = temp;
                } else {
                    x_in[bx * n_samples + ( n + unload_size )] = temp;
                }
            }
        }
    }
}

extern "C" __global__ void __launch_bounds__( 1024 ) _cupy_sosfilt_float32( const int n_signals,
                                                                            const int n_samples,
                                                                            const int n_sections,
                                                                            const int zi_width,
                                                                            const float *__restrict__ sos,
                                                                            const float *__restrict__ zi,
                                                                            float *__restrict__ x_in ) {

    extern __shared__ float s_buffer_f[];

    _cupy_sosfilt<float>( n_signals, n_samples, n_sections, zi_width, sos, zi, x_in, s_buffer_f );
}

extern "C" __global__ void __launch_bounds__( 1024 ) _cupy_sosfilt_float64( const int n_signals,
                                                                            const int n_samples,
                                                                            const int n_sections,
                                                                            const int zi_width,
                                                                            const double *__restrict__ sos,
                                                                            const double *__restrict__ zi,
                                                                            double *__restrict__ x_in ) {

    extern __shared__ double s_buffer_d[];

    _cupy_sosfilt<double>( n_signals, n_samples, n_sections, zi_width, sos, zi, x_in, s_buffer_d );
}
