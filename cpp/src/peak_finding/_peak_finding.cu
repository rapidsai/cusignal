
#include <hip/hip_runtime.h>
// Copyright (c) 2019-2020, NVIDIA CORPORATION.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


///////////////////////////////////////////////////////////////////////////////
//                            FUNCTION POINTERS                              //
///////////////////////////////////////////////////////////////////////////////


template<typename T>
__device__ __forceinline__ bool less( T a, T b ) {
    return ( a < b );
}

template<typename T>
__device__ __forceinline__ bool greater( T a, T b ) {
    return ( a > b );
}

template<typename T>
__device__ __forceinline__ bool less_equal( T a, T b ) {
    return ( a <= b );
}

template<typename T>
__device__ __forceinline__ bool greater_equal( T a, T b ) {
    return ( a >= b );
}

template<typename T>
__device__ __forceinline__ bool equal( T a, T b ) {
    return ( a == b );
}

template<typename T>
__device__ __forceinline__ bool not_equal( T a, T b ) {
    return ( a != b );
}

using op_func_i                      = bool ( * )( int, int );
__device__ op_func_i const func_i[6] = { less, greater, less_equal, greater_equal, equal, not_equal };

using op_func_l                      = bool ( * )( long int, long int );
__device__ op_func_l const func_l[6] = { less, greater, less_equal, greater_equal, equal, not_equal };

using op_func_f                      = bool ( * )( float, float );
__device__ op_func_f const func_f[6] = { less, greater, less_equal, greater_equal, equal, not_equal };

using op_func_d                      = bool ( * )( double, double );
__device__ op_func_d const func_d[6] = { less, greater, less_equal, greater_equal, equal, not_equal };

///////////////////////////////////////////////////////////////////////////////
//                          BOOLRELEXTREMA 1D                                //
///////////////////////////////////////////////////////////////////////////////

template<typename T, class U>
__device__ void _cupy_boolrelextrema_1D( const int  n,
                                      const int  order,
                                      const bool clip,
                                      const T *__restrict__ inp,
                                      bool *__restrict__ results,
                                      U func ) {

    const int tx { static_cast<int>( blockIdx.x * blockDim.x + threadIdx.x ) };
    const int stride { static_cast<int>( blockDim.x * gridDim.x ) };

    int plus {};
    int minus {};

    for ( int tid = tx; tid < n; tid += stride ) {

        T    data { inp[tid] };
        bool temp { true };

        for ( int o = 1; o < ( order + 1 ); o++ ) {
            plus  = tid + o;
            minus = tid - o;

            if ( clip ) {
                if ( plus >= n ) {
                    plus = n - 1;
                }
                if ( minus < 0 ) {
                    minus = 0;
                }
            } else {
                if ( plus >= n ) {
                    plus -= n;
                }
                if ( minus < 0 ) {
                    minus += n;
                }
            }
            temp &= func( data, inp[plus] );
            temp &= func( data, inp[minus] );
        }
        results[tid] = temp;
    }
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_1D_int32( const int  n,
                                                                                const int  order,
                                                                                const bool clip,
                                                                                const int  comp,
                                                                                const int *__restrict__ inp,
                                                                                bool *__restrict__ results ) {
    _cupy_boolrelextrema_1D<int, op_func_i>( n, order, clip, inp, results, func_i[comp] );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_1D_int64( const int  n,
                                                                                const int  order,
                                                                                const bool clip,
                                                                                const int  comp,
                                                                                const long int *__restrict__ inp,
                                                                                bool *__restrict__ results ) {
    _cupy_boolrelextrema_1D<long int, op_func_l>( n, order, clip, inp, results, func_l[comp] );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_1D_float32( const int  n,
                                                                                  const int  order,
                                                                                  const bool clip,
                                                                                  const int  comp,
                                                                                  const float *__restrict__ inp,
                                                                                  bool *__restrict__ results ) {
    _cupy_boolrelextrema_1D<float, op_func_f>( n, order, clip, inp, results, func_f[comp] );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_1D_float64( const int  n,
                                                                                  const int  order,
                                                                                  const bool clip,
                                                                                  const int  comp,
                                                                                  const double *__restrict__ inp,
                                                                                  bool *__restrict__ results ) {
    _cupy_boolrelextrema_1D<double, op_func_d>( n, order, clip, inp, results, func_d[comp] );
}

///////////////////////////////////////////////////////////////////////////////
//                          BOOLRELEXTREMA 2D                                //
///////////////////////////////////////////////////////////////////////////////

template<typename T, class U>
__device__ void _cupy_boolrelextrema_2D( const int  n,
                                      const int  order,
                                      const bool clip,
                                      const T *__restrict__ inp,
                                      bool *__restrict__ results,
                                      U func ) {

    const int tx { static_cast<int>( blockIdx.x * blockDim.x + threadIdx.x ) };
    const int stride { static_cast<int>( blockDim.x * gridDim.x ) };

    int plus {};
    int minus {};

    for ( int tid = tx; tid < n; tid += stride ) {

        T    data { inp[tid] };
        bool temp { true };

        for ( int o = 1; o < ( order + 1 ); o++ ) {
            plus  = tid + o;
            minus = tid - o;

            if ( clip ) {
                if ( plus >= n ) {
                    plus = n - 1;
                }
                if ( minus < 0 ) {
                    minus = 0;
                }
            } else {
                if ( plus >= n ) {
                    plus -= n;
                }
                if ( minus < 0 ) {
                    minus += n;
                }
            }
            temp &= func( data, inp[plus] );
            temp &= func( data, inp[minus] );
        }
        results[tid] = temp;
    }
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_2D_int32( const int  n,
                                                                                const int  order,
                                                                                const bool clip,
                                                                                const int  comp,
                                                                                const int *__restrict__ inp,
                                                                                bool *__restrict__ results ) {
    _cupy_boolrelextrema_2D<int, op_func_i>( n, order, clip, inp, results, func_i[comp] );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_2D_int64( const int  n,
                                                                                const int  order,
                                                                                const bool clip,
                                                                                const int  comp,
                                                                                const long int *__restrict__ inp,
                                                                                bool *__restrict__ results ) {
    _cupy_boolrelextrema_2D<long int, op_func_l>( n, order, clip, inp, results, func_l[comp] );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_2D_float32( const int  n,
                                                                                  const int  order,
                                                                                  const bool clip,
                                                                                  const int  comp,
                                                                                  const float *__restrict__ inp,
                                                                                  bool *__restrict__ results ) {
    _cupy_boolrelextrema_2D<float, op_func_f>( n, order, clip, inp, results, func_f[comp] );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_2D_float64( const int  n,
                                                                                  const int  order,
                                                                                  const bool clip,
                                                                                  const int  comp,
                                                                                  const double *__restrict__ inp,
                                                                                  bool *__restrict__ results ) {
    _cupy_boolrelextrema_2D<double, op_func_d>( n, order, clip, inp, results, func_d[comp] );
}