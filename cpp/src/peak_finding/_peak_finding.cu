
#include <hip/hip_runtime.h>
// Copyright (c) 2019-2020, NVIDIA CORPORATION.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

///////////////////////////////////////////////////////////////////////////////
//                            FUNCTION POINTERS                              //
///////////////////////////////////////////////////////////////////////////////

template<typename T>
__device__ __forceinline__ bool less( T a, T b ) {
    return ( a < b );
}

template<typename T>
__device__ __forceinline__ bool greater( T a, T b ) {
    return ( a > b );
}

template<typename T>
__device__ __forceinline__ bool less_equal( T a, T b ) {
    return ( a <= b );
}

template<typename T>
__device__ __forceinline__ bool greater_equal( T a, T b ) {
    return ( a >= b );
}

template<typename T>
__device__ __forceinline__ bool equal( T a, T b ) {
    return ( a == b );
}

template<typename T>
__device__ __forceinline__ bool not_equal( T a, T b ) {
    return ( a != b );
}

using op_func_i                      = bool ( * )( int, int );
__device__ op_func_i const func_i[6] = { less, greater, less_equal, greater_equal, equal, not_equal };

using op_func_l                      = bool ( * )( long int, long int );
__device__ op_func_l const func_l[6] = { less, greater, less_equal, greater_equal, equal, not_equal };

using op_func_f                      = bool ( * )( float, float );
__device__ op_func_f const func_f[6] = { less, greater, less_equal, greater_equal, equal, not_equal };

using op_func_d                      = bool ( * )( double, double );
__device__ op_func_d const func_d[6] = { less, greater, less_equal, greater_equal, equal, not_equal };

template<typename T>
__device__ __forceinline__ void clip_plus( const bool &clip, const int &n, T &plus ) {
    if ( clip ) {
        if ( plus >= n ) {
            plus = n - 1;
        }
    } else {
        if ( plus >= n ) {
            plus -= n;
        }
    }
}

template<typename T>
__device__ __forceinline__ void clip_minus( const bool &clip, const int &n, T &minus ) {
    if ( clip ) {
        if ( minus < 0 ) {
            minus = 0;
        }
    } else {
        if ( minus < 0 ) {
            minus += n;
        }
    }
}

///////////////////////////////////////////////////////////////////////////////
//                          BOOLRELEXTREMA 1D                                //
///////////////////////////////////////////////////////////////////////////////

template<typename T, class U>
__device__ void _cupy_boolrelextrema_1D( const int  n,
                                         const int  order,
                                         const bool clip,
                                         const T *__restrict__ inp,
                                         bool *__restrict__ results,
                                         U func ) {

    const int tx { static_cast<int>( blockIdx.x * blockDim.x + threadIdx.x ) };
    const int stride { static_cast<int>( blockDim.x * gridDim.x ) };

    for ( int tid = tx; tid < n; tid += stride ) {

        T    data { inp[tid] };
        bool temp { true };

        for ( int o = 1; o < ( order + 1 ); o++ ) {
            int plus { tid + o };
            int minus { tid - o };

            clip_plus( clip, n, plus );
            clip_minus( clip, n, minus );

            temp &= func( data, inp[plus] );
            temp &= func( data, inp[minus] );
        }
        results[tid] = temp;
    }
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_1D_int32( const int  n,
                                                                                   const int  order,
                                                                                   const bool clip,
                                                                                   const int  comp,
                                                                                   const int *__restrict__ inp,
                                                                                   bool *__restrict__ results ) {
    _cupy_boolrelextrema_1D<int, op_func_i>( n, order, clip, inp, results, func_i[comp] );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_1D_int64( const int  n,
                                                                                   const int  order,
                                                                                   const bool clip,
                                                                                   const int  comp,
                                                                                   const long int *__restrict__ inp,
                                                                                   bool *__restrict__ results ) {
    _cupy_boolrelextrema_1D<long int, op_func_l>( n, order, clip, inp, results, func_l[comp] );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_1D_float32( const int  n,
                                                                                     const int  order,
                                                                                     const bool clip,
                                                                                     const int  comp,
                                                                                     const float *__restrict__ inp,
                                                                                     bool *__restrict__ results ) {
    _cupy_boolrelextrema_1D<float, op_func_f>( n, order, clip, inp, results, func_f[comp] );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_1D_float64( const int  n,
                                                                                     const int  order,
                                                                                     const bool clip,
                                                                                     const int  comp,
                                                                                     const double *__restrict__ inp,
                                                                                     bool *__restrict__ results ) {
    _cupy_boolrelextrema_1D<double, op_func_d>( n, order, clip, inp, results, func_d[comp] );
}

///////////////////////////////////////////////////////////////////////////////
//                          BOOLRELEXTREMA 2D                                //
///////////////////////////////////////////////////////////////////////////////

template<typename T, class U>
__device__ void _cupy_boolrelextrema_2D( const int  in_x,
                                         const int  in_y,
                                         const int  order,
                                         const bool clip,
                                         const int  axis,
                                         const T *__restrict__ inp,
                                         bool *__restrict__ results,
                                         U func ) {

    const int ty { static_cast<int>( blockIdx.x * blockDim.x + threadIdx.x ) };
    const int tx { static_cast<int>( blockIdx.y * blockDim.y + threadIdx.y ) };

    if ( ( tx < in_y ) && ( ty < in_x ) ) {
        int tid { tx * in_x + ty };

        T    data { inp[tid] };
		bool temp { true };
		
        for ( int o = 1; o < ( order + 1 ); o++ ) {

            int plus {};
            int minus {};

            if ( axis == 0 ) {
                plus  = tx + o;
				minus = tx - o;

                clip_plus( clip, in_y, plus );
				clip_minus( clip, in_y, minus );
				
                plus  = plus * in_x + ty;
                minus = minus * in_x + ty;
            } else {
                plus  = ty + o;
                minus = ty - o;

                clip_plus( clip, in_x, plus );
                clip_minus( clip, in_x, minus );

                plus  = tx * in_x + plus;
                minus = tx * in_x + minus;
			}

            temp &= func( data, inp[plus] );
            temp &= func( data, inp[minus] );
        }
        results[tid] = temp;
    }
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_2D_int32( const int  in_x,
                                                                                   const int  in_y,
                                                                                   const int  order,
                                                                                   const bool clip,
                                                                                   const int  comp,
                                                                                   const int  axis,
                                                                                   const int *__restrict__ inp,
                                                                                   bool *__restrict__ results ) {
    _cupy_boolrelextrema_2D<int, op_func_i>( in_x, in_y, order, clip, axis, inp, results, func_i[comp] );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_2D_int64( const int  in_x,
                                                                                   const int  in_y,
                                                                                   const int  order,
                                                                                   const bool clip,
                                                                                   const int  comp,
                                                                                   const int  axis,
                                                                                   const long int *__restrict__ inp,
                                                                                   bool *__restrict__ results ) {
    _cupy_boolrelextrema_2D<long int, op_func_l>( in_x, in_y, order, clip, axis, inp, results, func_l[comp] );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_2D_float32( const int  in_x,
                                                                                     const int  in_y,
                                                                                     const int  order,
                                                                                     const bool clip,
                                                                                     const int  comp,
                                                                                     const int  axis,
                                                                                     const float *__restrict__ inp,
                                                                                     bool *__restrict__ results ) {
    _cupy_boolrelextrema_2D<float, op_func_f>( in_x, in_y, order, clip, axis, inp, results, func_f[comp] );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_boolrelextrema_2D_float64( const int  in_x,
                                                                                     const int  in_y,
                                                                                     const int  order,
                                                                                     const bool clip,
                                                                                     const int  comp,
                                                                                     const int  axis,
                                                                                     const double *__restrict__ inp,
                                                                                     bool *__restrict__ results ) {
    _cupy_boolrelextrema_2D<double, op_func_d>( in_x, in_y, order, clip, axis, inp, results, func_d[comp] );
}