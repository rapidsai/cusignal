#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <thrust/complex.h>

///////////////////////////////////////////////////////////////////////////////
//                                READER                                     //
///////////////////////////////////////////////////////////////////////////////

// Byte swap short
__device__ short swap_int16( short val ) {
    return ( val << 8 ) | ( ( val >> 8 ) & 0xFF );
}

// Byte swap unsigned short
__device__ unsigned short swap_uint16( unsigned short val ) {
    return ( val << 8 ) | ( val >> 8 );
}

// Byte swap int
__device__ int swap_int32( int val ) {
    val = ( ( val << 8 ) & 0xFF00FF00 ) | ( ( val >> 8 ) & 0xFF00FF );
    return ( val << 16 ) | ( ( val >> 16 ) & 0xFFFF );
}

// Byte swap unsigned int
__device__ unsigned int swap_uint32( unsigned int val ) {
    val = ( ( val << 8 ) & 0xFF00FF00 ) | ( ( val >> 8 ) & 0xFF00FF );
    return ( val << 16 ) | ( val >> 16 );
}

// Byte swap float
__device__ float swap_float( float val ) {
    float retVal;
    char *floatToConvert = reinterpret_cast<char *>( &val );
    char *returnFloat    = reinterpret_cast<char *>( &retVal );

    int ds = sizeof( float );  // data size

// swap the bytes into a temporary buffer
#pragma unroll 4
    for ( int i = 0; i < ds; i++ ) {
        returnFloat[i] = floatToConvert[( ds - 1 ) - i];
    }

    return retVal;
}

__device__ double swap_double( double val ) {
    double retVal;
    char * doubleToConvert = reinterpret_cast<char *>( &val );
    char * returnDouble    = reinterpret_cast<char *>( &retVal );

    int ds = sizeof( double );  // data size

// swap the bytes into a temporary buffer
#pragma unroll 8
    for ( int i = 0; i < ds; i++ ) {
        returnDouble[i] = doubleToConvert[( ds - 1 ) - i];
    }

    return retVal;
}

template<typename T>
__device__ void
_cupy_unpack( const size_t N, const bool little, unsigned char *__restrict__ input, T *__restrict__ output ) {

    const int tx { static_cast<int>( blockIdx.x * blockDim.x + threadIdx.x ) };
    const int stride { static_cast<int>( blockDim.x * gridDim.x ) };

    for ( int tid = tx; tid < N; tid += stride ) {

        if ( little ) {
            output[tid] = reinterpret_cast<T *>( input )[tid];
        } else {
            T data = reinterpret_cast<T *>( input )[tid];

#if __cplusplus >= 201703L
            if constexpr ( std::is_same<T, char>::value || std::is_same<T, unsigned char>::value ) {
                output[tid] = data;
            } else if constexpr ( std::is_same<T, short>::value ) {
                output[tid] = swap_int16( data );
            } else if constexpr ( std::is_same<T, unsigned short>::value ) {
                output[tid] = swap_uint16( data );
            } else if constexpr ( std::is_same<T, int>::value ) {
                output[tid] = swap_int32( data );
            } else if constexpr ( std::is_same<T, unsigned int>::value ) {
                output[tid] = swap_uint32( data );
            } else if constexpr ( std::is_same<T, float>::value ) {
                output[tid] = swap_float( data );
            } else if constexpr ( std::is_same<T, double>::value ) {
                output[tid] = swap_double( data );
            } else if constexpr ( std::is_same<T, thrust::complex<float>>::value ) {
                float real  = swap_float( data.real( ) );
                float imag  = swap_float( data.imag( ) );
                output[tid] = thrust::complex<float>( real, imag );
            } else if constexpr ( std::is_same<T, thrust::complex<double>>::value ) {
                double real = swap_double( data.real( ) );
                double imag = swap_double( data.imag( ) );
                output[tid] = thrust::complex<double>( real, imag );
            }
#else
            if ( std::is_same<T, char>::value ) {
                output[tid] = data;
            } else if ( std::is_same<T, short>::value ) {
                output[tid] = swap_int16( data );
            } else if ( std::is_same<T, unsigned short>::value ) {
                output[tid] = swap_uint16( data );
            } else if ( std::is_same<T, int>::value ) {
                output[tid] = swap_int32( data );
            } else if ( std::is_same<T, unsigned int>::value ) {
                output[tid] = swap_uint32( data );
            } else if ( std::is_same<T, float>::value ) {
                output[tid] = swap_float( data );
            } else if ( std::is_same<T, double>::value ) {
                output[tid] = swap_double( data );
            }
#endif
        }
    }
}

#if __cplusplus < 201703L
template<typename T>
__device__ void
_cupy_unpack_complex( const size_t N, const bool little, unsigned char *__restrict__ input, T *__restrict__ output ) {

    const int tx { static_cast<int>( blockIdx.x * blockDim.x + threadIdx.x ) };
    const int stride { static_cast<int>( blockDim.x * gridDim.x ) };

    for ( int tid = tx; tid < N; tid += stride ) {

        if ( little ) {
            output[tid] = reinterpret_cast<T *>( input )[tid];
        } else {
            T data = reinterpret_cast<T *>( input )[tid];

            if ( std::is_same<T, thrust::complex<float>>::value ) {
                float real = swap_float( data.real( ) );
                float imag = swap_float( data.imag( ) );

                output[tid] = thrust::complex<float>( real, imag );
            } else if ( std::is_same<T, thrust::complex<double>>::value ) {
                double real = swap_double( data.real( ) );
                double imag = swap_double( data.imag( ) );

                output[tid] = thrust::complex<double>( real, imag );
            }
        }
    }
}
#endif

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_unpack_int8( const size_t N,
                                                                       const bool   little,
                                                                       unsigned char *__restrict__ input,
                                                                       char *__restrict__ output ) {
    _cupy_unpack<char>( N, little, input, output );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_unpack_uint8( const size_t N,
                                                                        const bool   little,
                                                                        unsigned char *__restrict__ input,
                                                                        unsigned char *__restrict__ output ) {
    _cupy_unpack<unsigned char>( N, little, input, output );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_unpack_int16( const size_t N,
                                                                        const bool   little,
                                                                        unsigned char *__restrict__ input,
                                                                        short *__restrict__ output ) {
    _cupy_unpack<short>( N, little, input, output );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_unpack_uint16( const size_t N,
                                                                         const bool   little,
                                                                         unsigned char *__restrict__ input,
                                                                         unsigned short *__restrict__ output ) {
    _cupy_unpack<unsigned short>( N, little, input, output );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_unpack_int32( const size_t N,
                                                                        const bool   little,
                                                                        unsigned char *__restrict__ input,
                                                                        int *__restrict__ output ) {
    _cupy_unpack<int>( N, little, input, output );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_unpack_uint32( const size_t N,
                                                                         const bool   little,
                                                                         unsigned char *__restrict__ input,
                                                                         unsigned int *__restrict__ output ) {
    _cupy_unpack<unsigned int>( N, little, input, output );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_unpack_float32( const size_t N,
                                                                          const bool   little,
                                                                          unsigned char *__restrict__ input,
                                                                          float *__restrict__ output ) {
    _cupy_unpack<float>( N, little, input, output );
}

extern "C" __global__ void __launch_bounds__( 512 ) _cupy_unpack_float64( const size_t N,
                                                                          const bool   little,
                                                                          unsigned char *__restrict__ input,
                                                                          double *__restrict__ output ) {
    _cupy_unpack<double>( N, little, input, output );
}

#if __cplusplus >= 201703L
extern "C" __global__ void __launch_bounds__( 512 )
    _cupy_unpack_complex64( const size_t N,
                            const bool   little,
                            unsigned char *__restrict__ input,
                            thrust::complex<float> *__restrict__ output ) {
    _cupy_unpack<thrust::complex<float>>( N, little, input, output );
}

extern "C" __global__ void __launch_bounds__( 512 )
    _cupy_unpack_complex128( const size_t N,
                             const bool   little,
                             unsigned char *__restrict__ input,
                             thrust::complex<double> *__restrict__ output ) {
    _cupy_unpack<thrust::complex<double>>( N, little, input, output );
}
#else
extern "C" __global__ void __launch_bounds__( 512 )
    _cupy_unpack_complex64( const size_t N,
                            const bool little,
                            unsigned char *__restrict__ input,
                            thrust::complex<float> *__restrict__ output ) {
    _cupy_unpack_complex<thrust::complex<float>>( N, little, input, output );
}

extern "C" __global__ void __launch_bounds__( 512 )
    _cupy_unpack_complex128( const size_t N,
                             const bool little,
                             unsigned char *__restrict__ input,
                             thrust::complex<double> *__restrict__ output ) {
    _cupy_unpack_complex<thrust::complex<double>>( N, little, input, output );
}
#endif